//header files
//standard headers 
#include<stdio.h>

//cuda headers
#include<hip/hip_runtime.h>

//global variables
const int iNumbersOfArrayElements = 5;

float *hostInput1 = NULL;
float *hostInput2 = NULL;
float *hostOutput = NULL;

float *deviceInput1 = NULL;
float *deviceInput2 = NULL;
float *deviceOutput = NULL;

// cuda kernel
__global__ void vecAddGPU(float *in1, float *in2, float *out, int len)
{
	//code
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

// entry point function
int main(void)
{
	//function declarations
	void cleanup(void);

	//variable declarations
	int size = iNumbersOfArrayElements * sizeof(float);
	hipError_t result = hipSuccess;

	// code
	// host memory allocation
	hostInput1 = (float *)malloc(size);
	if (hostInput1 == NULL)
	{
		printf("Host Memory Allocation Is Failed For hostInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float *)malloc(size);
	if (hostInput2 == NULL)
	{
		printf("Host Memory Allocation Is Failed For hostInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostOutput = (float *)malloc(size);
	if (hostOutput == NULL)
	{
		printf("Host Memory Allocation Is Failed For hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// filling values into host arrays
	hostInput1[0] = 101.0f;
	hostInput1[1] = 102.0f;
	hostInput1[2] = 103.0f;
	hostInput1[3] = 104.0f;
	hostInput1[4] = 105.0f;

	hostInput2[0] = 201.0f;
	hostInput2[1] = 202.0f;
	hostInput2[2] = 203.0f;
	hostInput2[3] = 204.0f;
	hostInput2[4] = 205.0f;

	//device memory allocation 
	result = hipMalloc((void **)&deviceInput1, size);
	if (result != hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&deviceInput2, size);
	if (result != hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&deviceOutput, size);
	if (result != hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// copy data from host arrays into device arrays
	result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to Device data copy is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to Device data copy is failed for deviceInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	dim3 dimGrid = dim3(iNumbersOfArrayElements, 1, 1);
	dim3 dimBlock = dim3(1, 1, 1);

	// cuda kernel for vector addition
	vecAddGPU <<<dimGrid, dimBlock >>> (deviceInput1, deviceInput2, deviceOutput, iNumbersOfArrayElements);

	// copy data from device array into host array
	result = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		printf("Device to Host Data Copy is failed for hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// vector addition on host 
	for (int i = 0; i < iNumbersOfArrayElements; i++)
	{
		printf("%f + %f = %f\n", hostInput1[i], hostInput2[i], hostOutput[i]);
	}

	//cleanup
	cleanup();

	return(0);
}

void cleanup(void)
{
	//code
	if (deviceOutput)
	{
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	if (deviceInput2)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	if (hostOutput)
	{
		hipFree(hostOutput);
		hostOutput = NULL;
	}

	if (hostInput2)
	{
		hipFree(hostInput2);
		hostInput2 = NULL;
	}

	if (hostInput1)
	{
		hipFree(hostInput1);
		hostInput1 = NULL;
	}
}

